#include "hip/hip_runtime.h"
#include "adder.cuh"

__global__ void adder(int *a, int *b, int *res, int count)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	while(tid < count)
	{
		res[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}

	return;
}

void adder(int *a, int *b, int *res, int count, int threadcount, int blockcount)
{

	int *a_d, *b_d, *res_d;
	hipMalloc((void **)&a_d, sizeof(int)*count);
	hipMalloc((void **)&b_d, sizeof(int)*count);
	hipMalloc((void **)&res_d, sizeof(int)*count);

	hipMemcpy(a_d, a, sizeof(int)*count, hipMemcpyHostToDevice);
	hipMemcpy(b_d, b, sizeof(int)*count, hipMemcpyHostToDevice);
	adder<<<threadcount, blockcount>>>(a_d, b_d, res_d, count);
	
	hipMemcpy(res, res_d, sizeof(int)*count, hipMemcpyDeviceToHost);

	return;
}
